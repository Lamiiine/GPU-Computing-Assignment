#include<stdio.h>
#include<hip/hip_runtime.h>

__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
int i = (threadIdx.x + blockDim.x * blockIdx.x)*2;
if(i<n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n)
{
int size = n * sizeof(float);
float *d_A, *d_B, *d_C ;

hipMalloc((void**) &d_A, size);
hipMalloc((void**) &d_B, size);

hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
hipMemcpy(d_B,B, size, hipMemcpyHostToDevice);
hipMalloc((void**) &d_C, size);

vecAddKernel<<<ceil(n/256.0),256>>>(d_A, d_B, d_C, n);
hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

hipFree(d_A);hipFree(d_B);hipFree(d_C);
}


int main()
{
float *h_A= new float[5], *h_B= new float[5], *h_C = new float[5];
for(int i = 0; i < 5 ; i++)h_A[i]=1,h_B[i]=3;

vecAdd(h_A,h_B,h_C,5);

for(int i = 0 ; i< 5;i++) printf("%f\n", h_C[i]);
return 0;
}








