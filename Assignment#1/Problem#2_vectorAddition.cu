#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>

__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
int i = (threadIdx.x + blockDim.x * blockIdx.x)*2;
if(i<n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n)
{
int size = n * sizeof(float);
float *d_A, *d_B, *d_C ;
//Allocating memory on device
hipMalloc((void**) &d_A, size);
hipMalloc((void**) &d_B, size);
  
//Copy data from Host to Device
hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
hipMemcpy(d_B,B, size, hipMemcpyHostToDevice);
  
//Allocating memory for the output  
hipMalloc((void**) &d_C, size);

vecAddKernel<<<ceil(n/256.0),256>>>(d_A, d_B, d_C, n);
//Copy data from Device to Host
hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

//Free memory in Device
hipFree(d_A);hipFree(d_B);hipFree(d_C);
}


int main()
{
  int n;
  //Size of the vectors
  scanf("%d", &n);
  
  //Allocating memory on Host
  float *h_A= new float[n], *h_B= new float[n], *h_C = new float[n];
  
  //Initializing vectors with random values
  srand(time(NULL));
	for (int i = 0; i < n; i++)
	{
		h_A[i] = rand(); h_B[i] = rand();
	}
  
  vecAdd(h_A,h_B,h_C,n);

  for(int i = 0 ; i< n ;i++) printf("%f\n", h_C[i]);
  
return 0;
}








